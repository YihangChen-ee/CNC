#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/NumericUtils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAGeneratorImpl.h>
#include <ATen/cuda/CUDAGraphsUtils.cuh>
#include <c10/util/MaybeOwned.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>

#include "include/data_spec.hpp"
#include "include/data_spec_packed.cuh"
#include "include/utils_cuda.cuh"
#include "include/utils_grid.cuh"
#include "include/utils_math.cuh"

static constexpr uint32_t MAX_GRID_LEVELS = 8;

namespace {
namespace device {

inline __device__ float _calc_dt(
    const float t, const float cone_angle,
    const float dt_min, const float dt_max)
{
    return clamp(t * cone_angle, dt_min, dt_max);
}

/* Ray traversal within multiple voxel grids. 

About rays:
    Each ray is defined by its origin (rays_o) and unit direction (rays_d). We also allows
    a optional boolen ray mask (rays_mask) to indicate whether we want to skip some rays. 

About voxel grids:
    We support ray traversal through one or more voxel grids (n_grids). Each grid is defined
    by an axis-aligned AABB (aabbs), and a binary occupancy grid (binaries) with resolution of
    {resx, resy, resz}. Currently, we assume all grids have the same resolution. Note the ordering
    of the grids is important when there are overlapping grids, because we assume the grid in front
    has higher priority when examing occupancy status (e.g., the first grid's occupancy status
    will overwrite the second grid's occupancy status if they overlap).

About ray grid intersections:
    We require the ray grid intersections to be precomputed and sorted. Specifically, if hit, each 
    ray-grid pair has two intersections, one for entering the grid and one for leaving the grid. 
    For multiple grids, there are in total 2 * n_grids intersections for each ray. The intersections
    are sorted by the distance to the ray origin (t_sorted). We take a boolen array (hits) to indicate 
    whether each ray-grid pair is hit. We also need a int64 array (t_indices) to indicate the grid id
    (0-index) for each intersection.

About ray traversal:
    The ray is traversed through the grids in the order of the sorted intersections. We allows pre-ray
    near and far planes (near_planes, far_planes) to be specified. Early termination can be controlled by
    setting the maximum traverse steps via traverse_steps_limit. We also allow an optional step size
    (step_size) to be specified. If step_size <= 0.0, we will record the steps of the ray pass through
    each voxel cell. Otherwise, we will use the step_size to march through the grids. When step_size > 0.0,
    we also allow a cone angle (cone_angle) to be provides, to linearly increase the step size as the ray
    goes further away from the origin (see _calc_dt()). cone_angle should be always >= 0.0, and 0.0 
    means uniform marching with step_size.

About outputs:
    The traversal intervals and samples are stored in `intervals` and `samples` respectively. Additionally,
    we also return where the traversal actually terminates (terminate_planes). This is useful when 
    traverse_steps_limit is set (traverse_steps_limit > 0) as the ray may not reach the far plane or the
    boundary of the grids.
*/
__global__ void traverse_grids_kernel(
    // rays
    int32_t n_rays,
    float *rays_o,  // [n_rays, 3]
    float *rays_d,  // [n_rays, 3]
    bool *rays_mask, // [n_rays]
    // grids
    int32_t n_grids,
    int3 resolution,
    bool *binaries, // [n_grids, resx, resy, resz]
    float *aabbs,   // [n_grids, 6] [[-1.5000, -1.5000, -1.5000,  1.5000,  1.5000,  1.5000]]
    // sorted intersections
    bool *hits,         // [n_rays, n_grids]
    float *t_sorted,    // [n_rays, n_grids * 2]  // 这里代表的是xyz中的最快轴
    int64_t *t_indices, // [n_rays, n_grids * 2]
    // options
    float *near_planes,  // [n_rays] 对于blender，是0
    float *far_planes,   // [n_rays] 对于blender，是1e10
    float step_size,  // 默认是1e-3
    float cone_angle,
    int32_t traverse_steps_limit,
    // outputs
    bool first_pass,
    PackedRaySegmentsSpec intervals,
    PackedRaySegmentsSpec samples,
    float *terminate_planes)
{
    float eps = 1e-6f;

    // parallelize over rays
    for (int32_t tid = blockIdx.x * blockDim.x + threadIdx.x; tid < n_rays; tid += blockDim.x * gridDim.x)
    {
        if (rays_mask != nullptr && !rays_mask[tid]) continue;

        // skip rays that are empty.
        if (intervals.chunk_cnts != nullptr)
            if (!first_pass && intervals.chunk_cnts[tid] == 0) continue;
        if (samples.chunk_cnts != nullptr)
            if (!first_pass && samples.chunk_cnts[tid] == 0) continue;

        int64_t chunk_start, chunk_start_bin;
        if (!first_pass) {
            if (intervals.chunk_cnts != nullptr)
                chunk_start = intervals.chunk_starts[tid];
            if (samples.chunk_cnts != nullptr)
                chunk_start_bin = samples.chunk_starts[tid];
        }
        float near_plane = near_planes[tid];
        float far_plane = far_planes[tid];

        // 这边是指针。*3是因为3维。对每一条光线进行遍历。
        SingleRaySpec ray = SingleRaySpec(
            rays_o + tid * 3, rays_d + tid * 3, near_plane, far_plane);

        int32_t base_hits = tid * n_grids;
        // 变量 base_t_sorted 表示用于访问 t_sorted 数组的特定射线的起始索引。
        // t_sorted 数组包含每条射线与每个网格的排序交点距离，其大小为 [n_rays, n_grids * 2]。每一行对应于一条特定的射线，每一对列表示沿射线路径进入和离开每个网格的交点距离。
        // 变量 base_t_sorted 的计算方式是 tid * n_grids * 2，其中 tid 表示线程ID。它作为访问特定射线的 t_sorted 数组的起始索引。
        // 通过使用 base_t_sorted，代码可以通过从 base_t_sorted 到 base_t_sorted + n_grids * 2 - 1 的索引范围内访问特定射线的交点距离。这样，代码就能够遍历特定射线的排序交点，并在遍历过程中进行相应的处理。
        int32_t base_t_sorted = tid * n_grids * 2;

        // loop over all intersections along the ray.
        // 这里开始循环遍历沿光线的交点，这些交点是按距离排序的。
        int64_t n_intervals = 0;
        int64_t n_samples = 0;
        float t_last = near_plane;
        // continuous: means whether current traverse point is continuous with both its previous and next point along the ray.
        // it will be false if 1. it is the first 
        bool continuous = false;
        // 对grid进行遍历。对于blender，只有一个grid
        for (int32_t i = base_t_sorted; i < base_t_sorted + n_grids * 2 - 1; i++) {
            // whether this is the entering or leaving for this level of grid.
            bool is_entering = t_indices[i] < n_grids;
            int64_t level = t_indices[i] % n_grids;
            // printf("i=%d, level=%lld, is_entering=%d, hits=%d\n", i, level, is_entering, hits[level]);

            // 在对每个grid的循环迭代中，检查当前ray是否在grid内，如果不在grid内，则跳过。
            if (!hits[base_hits + level]) {
                continue; // this grid is not hit.
            }

            // 如果当前交点不是进入网格的点（即，它是离开网格的点），则检查下一个交点是否进入另一个网格。如果是，说明光线在网格之间穿越，继续下一轮循环。
            if (!is_entering) {
                // we are leaving this grid. Are we inside the next grid?
                bool next_is_entering = t_indices[i + 1] < n_grids;
                if (next_is_entering) continue; // we are outside next grid.
                level = t_indices[i + 1] % n_grids;
                if (!hits[base_hits + level]) {
                    continue; // this grid is not hit.
                }
            }

            // 在每个循环迭代中，计算有效区间的最小和最大值。
            float this_tmin = fmaxf(t_sorted[i], near_plane);
            float this_tmax = fminf(t_sorted[i + 1], far_plane);   
            if (this_tmin >= this_tmax) continue; // this interval is invalid. e.g. (0.0f, 0.0f)
            // printf("i=%d, this_tmin=%f, this_tmax=%f, level=%lld\n", i, this_tmin, this_tmax, level);

            if (!continuous) {  // continuous: means whether current traverse point is continuous with both its previous and next point along the ray.
                if (step_size <= 0.0f) { // march to this_tmin.
                    t_last = this_tmin;
                } else {
                    // 对于blender，cone_angle=0，所以dt=step_size
                    float dt = _calc_dt(t_last, cone_angle, step_size, 1e10f);
                    while (true) { // march until t_mid is right after this_tmin.
                        if (t_last + dt * 0.5f >= this_tmin) break;
                        t_last += dt;  // 修改了t_last，为下一个xx做准备
                    }
                }
            }
            // printf(
            //     "[traverse segment] i=%d, this_mip=%d, this_tmin=%f, this_tmax=%f\n", 
            //     i, this_mip, this_tmin, this_tmax);

            AABBSpec aabb = AABBSpec(aabbs + level * 6);

            // init: pre-compute variables needed for traversal
            float3 tdist, delta;
            // tdist 中的每个分量（tdist.x、tdist.y 和 tdist.z）包含了光线在对应轴上到达下一个交点的距离信息。
            // delta 这个变量表示在体素网格中沿着光线方向的步进量
            int3 step_index, current_index, final_index;
            // get voxel index of start and end within grid
            setup_traversal(
                ray, this_tmin, this_tmax, eps,
                aabb, resolution,
                // outputs
                delta, tdist, step_index, current_index, final_index);
            // printf(
            //     "[traverse init], delta=(%f, %f, %f), step_index=(%d, %d, %d)\n",
            //     delta.x, delta.y, delta.z, step_index.x, step_index.y, step_index.z
            // );

            const int3 overflow_index = final_index + step_index;
            while (traverse_steps_limit <= 0 || n_samples < traverse_steps_limit) {
                // The code calculates t_traverse as the minimum value among the distances tdist.x, tdist.y, and tdist.z. 
                // These distances represent the distances to the next grid cell boundaries along the ray's path.
                float t_traverse = min(tdist.x, min(tdist.y, tdist.z)); // 和t_sorted一样，这里只考虑最快轴
                // The code then clamps t_traverse to ensure it does not exceed this_tmax, 
                // which represents the maximum intersection distance for the current grid cell.
                t_traverse = fminf(t_traverse, this_tmax);
                // resolution = make_int3(binaries.size(1), binaries.size(2), binaries.size(3));
                int64_t cell_id = (
                    current_index.x * resolution.y * resolution.z
                    + current_index.y * resolution.z
                    + current_index.z
                    + level * resolution.x * resolution.y * resolution.z
                );

                if (!binaries[cell_id]) {
                    // skip the cell that is empty.
                    if (step_size <= 0.0f) { // march to t_traverse.
                        t_last = t_traverse;
                    } else {
                        // 对于blender，cone_angle=0，所以dt=step_size
                        float dt = _calc_dt(t_last, cone_angle, step_size, 1e10f);
                        while (true) { // march until t_mid is right after t_traverse.
                            if (t_last + dt * 0.5f >= t_traverse) break;
                            t_last += dt;  // 修改了t_last，为下一个xx做准备
                        }
                    }
                    continuous = false;
                } else {
                    // this cell is not empty, so we need to traverse it.
                    // traverse_steps_limit: Maximum number of samples per ray. default: None
                    // 这个while表示在当前binary cell里进行工作。是否超出了当前cell是通过t_traverse来判断的。
                    // t_traverse是根据binary的resolution和rays_d得到的，在当前cell的最大遍历步长。
                    while (traverse_steps_limit <= 0 || n_samples < traverse_steps_limit) {
                        float t_next;
                        if (step_size <= 0.0f) {
                            t_next = t_traverse;
                        } else {  // march until t_mid is right after t_traverse.
                            float dt = _calc_dt(t_last, cone_angle, step_size, 1e10f);
                            if (t_last + dt * 0.5f >= t_traverse) break;  // 如果中点超出t_traverse了就跳出
                            t_next = t_last + dt;
                        }

                        /* 在循环中
                        idx:                    0   1   2   3
                        intervals.vals:         t0  t1  t2  t3
                        intervals.is_left:      T   T   T
                        intervals.is_right:         T   T   T
                        */

                        // writeout the interval.
                        if (intervals.chunk_cnts != nullptr) {
                            if (!continuous) {
                                if (!first_pass) {  // left side of the intervel
                                    int64_t idx = chunk_start + n_intervals;
                                    intervals.vals[idx] = t_last;  // 在外面的代码中会通过(t_origins + t_dirs * (t_starts[:, None] + t_ends[:, None]) / 2.0)转化成3D坐标
                                    intervals.ray_indices[idx] = tid;
                                    intervals.is_left[idx] = true;
                                }
                                n_intervals++;
                                if (!first_pass) {  // right side of the intervel
                                    int64_t idx = chunk_start + n_intervals;
                                    intervals.vals[idx] = t_next;  // 在外面的代码中会通过(t_origins + t_dirs * (t_starts[:, None] + t_ends[:, None]) / 2.0)转化成3D坐标
                                    intervals.ray_indices[idx] = tid;
                                    intervals.is_right[idx] = true;
                                }
                                n_intervals++;
                            } else {
                                if (!first_pass) {  // right side of the intervel
                                    int64_t idx = chunk_start + n_intervals;
                                    intervals.vals[idx] = t_next;  // 在外面的代码中会通过(t_origins + t_dirs * (t_starts[:, None] + t_ends[:, None]) / 2.0)转化成3D坐标
                                    intervals.ray_indices[idx] = tid;
                                    intervals.is_left[idx - 1] = true;
                                    intervals.is_right[idx] = true;
                                }
                                n_intervals++;
                            }
                        }

                        // writeout the sample.
                        if (samples.chunk_cnts != nullptr) {
                            if (!first_pass) {
                                int64_t idx = chunk_start_bin + n_samples;

                                samples.vals[idx] = (t_next + t_last) * 0.5f; // 实际计算的positions=(t_origins + t_dirs * (t_starts[:, None] + t_ends[:, None]) / 2.0)
                                // (t_next + t_last) * 0.5f = (t_last + dt + t_last) * 0.5f = t_last + dt*0.5 < t_traverse

                                samples.ray_indices[idx] = tid;
                                samples.is_valid[idx] = true;
                            }
                        }

                        n_samples++;
                        continuous = true;
                        t_last = t_next;
                        if (t_next >= t_traverse) break;
                    }
                }

                // printf(
                //     "[traverse], t_last=%f, t_traverse=%f, cell_id=%d, current_index=(%d, %d, %d)\n",
                //     t_last, t_traverse, cell_id, current_index.x, current_index.y, current_index.z
                // );

                if (!single_traversal(tdist, current_index, overflow_index, step_index, delta)) {
                    break;
                }
            }
        }
        if (terminate_planes != nullptr)
            terminate_planes[tid] = t_last;

        if (intervals.chunk_cnts != nullptr)
            intervals.chunk_cnts[tid] = n_intervals;
        if (samples.chunk_cnts != nullptr)
            samples.chunk_cnts[tid] = n_samples;
    }
}

__global__ void ray_aabb_intersect_kernel(
    const int32_t n_rays, float *rays_o, float *rays_d, float near, float far,
    const int32_t n_aabbs, float *aabbs,
    // outputs
    const float miss_value,
    float *t_mins, float *t_maxs, bool *hits)
{
    int32_t numel = n_rays * n_aabbs;
    // parallelize over rays
    for (int32_t tid = blockIdx.x * blockDim.x + threadIdx.x; tid < numel; tid += blockDim.x * gridDim.x)
    {
        int32_t ray_id = tid / n_aabbs;
        int32_t aabb_id = tid % n_aabbs;

        float t_min, t_max;
        bool hit = device::ray_aabb_intersect(
            SingleRaySpec(rays_o + ray_id * 3, rays_d + ray_id * 3, near, far), 
            AABBSpec(aabbs + aabb_id * 6), 
            t_min, t_max
        );
        if (hit) {   
            t_mins[tid] = t_min;
            t_maxs[tid] = t_max;
        } else {
            t_mins[tid] = miss_value;
            t_maxs[tid] = miss_value;
        }
        hits[tid] = hit;
    }
}


}  // namespace device
}  // namespace


std::tuple<RaySegmentsSpec, RaySegmentsSpec, torch::Tensor> traverse_grids(
    // rays
    const torch::Tensor rays_o, // [n_rays, 3]
    const torch::Tensor rays_d, // [n_rays, 3]
    const torch::Tensor rays_mask,   // [n_rays]
    // grids
    const torch::Tensor binaries,  // [n_grids, resx, resy, resz]
    const torch::Tensor aabbs,     // [n_grids, 6]  [[-1.5000, -1.5000, -1.5000,  1.5000,  1.5000,  1.5000]]
    // intersections
    const torch::Tensor t_sorted,  // [n_rays, n_grids]
    const torch::Tensor t_indices,  // [n_rays, n_grids]
    const torch::Tensor hits,    // [n_rays, n_grids]
    // options
    const torch::Tensor near_planes,
    const torch::Tensor far_planes,
    const float step_size, // 默认是1e-3
    const float cone_angle,
    const bool compute_intervals,
    const bool compute_samples,
    const bool compute_terminate_planes,
    const int32_t traverse_steps_limit, // <= 0 means no limit
    const bool over_allocate) // over allocate the memory for intervals and samples
{
    DEVICE_GUARD(rays_o);
    if (over_allocate) {
        TORCH_CHECK(traverse_steps_limit > 0, "traverse_steps_limit must be > 0 when over_allocate is true");
    }

    int32_t n_rays = rays_o.size(0);
    int32_t n_grids = binaries.size(0);
    int3 resolution = make_int3(binaries.size(1), binaries.size(2), binaries.size(3));

    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
    int32_t max_threads = 512; 
    int32_t max_blocks = 65535;
    dim3 threads = dim3(min(max_threads, n_rays));
    dim3 blocks = dim3(min(max_blocks, ceil_div<int32_t>(n_rays, threads.x)));

    // outputs
    RaySegmentsSpec intervals, samples;
    torch::Tensor terminate_planes;
    if (compute_terminate_planes) 
        terminate_planes = torch::empty({n_rays}, rays_o.options());

    if (over_allocate) {
        // over allocate the memory so that we can traverse the grids in a single pass.
        if (compute_intervals) {
            intervals.chunk_cnts = torch::full({n_rays}, traverse_steps_limit * 2, rays_o.options().dtype(torch::kLong)) * rays_mask;
            intervals.memalloc_data_from_chunk(true, true);
        }
        if (compute_samples) {
            samples.chunk_cnts = torch::full({n_rays}, traverse_steps_limit, rays_o.options().dtype(torch::kLong)) * rays_mask;
            samples.memalloc_data_from_chunk(false, true, true);
        }

        device::traverse_grids_kernel<<<blocks, threads, 0, stream>>>(
            // rays
            n_rays,
            rays_o.data_ptr<float>(),  // [n_rays, 3]
            rays_d.data_ptr<float>(),  // [n_rays, 3]
            rays_mask.data_ptr<bool>(),  // [n_rays]
            // grids
            n_grids,
            resolution,
            binaries.data_ptr<bool>(), // [n_grids, resx, resy, resz]
            aabbs.data_ptr<float>(),   // [n_grids, 6]
            // sorted intersections
            hits.data_ptr<bool>(),         // [n_rays, n_grids]
            t_sorted.data_ptr<float>(),    // [n_rays, n_grids * 2]
            t_indices.data_ptr<int64_t>(), // [n_rays, n_grids * 2]
            // options
            near_planes.data_ptr<float>(), // [n_rays]
            far_planes.data_ptr<float>(),  // [n_rays]
            step_size,
            cone_angle,
            traverse_steps_limit,
            // outputs
            false,
            device::PackedRaySegmentsSpec(intervals),
            device::PackedRaySegmentsSpec(samples),
            compute_terminate_planes ? terminate_planes.data_ptr<float>() : nullptr);
        
        // update the chunk starts with the actual chunk_cnts from traversal.
        intervals.compute_chunk_start();
        samples.compute_chunk_start();
    } else {
        // To allocate the accurate memory we need to traverse the grids twice.
        // The first pass is to count the number of segments along each ray.
        // The second pass is to fill the segments.
        if (compute_intervals)
            intervals.chunk_cnts = torch::empty({n_rays}, rays_o.options().dtype(torch::kLong));
        if (compute_samples)
            samples.chunk_cnts = torch::empty({n_rays}, rays_o.options().dtype(torch::kLong));
        device::traverse_grids_kernel<<<blocks, threads, 0, stream>>>(
            // rays
            n_rays,
            rays_o.data_ptr<float>(),  // [n_rays, 3]
            rays_d.data_ptr<float>(),  // [n_rays, 3]
            nullptr,  /* rays_mask */
            // grids
            n_grids,
            resolution,
            binaries.data_ptr<bool>(), // [n_grids, resx, resy, resz]
            aabbs.data_ptr<float>(),   // [n_grids, 6]
            // sorted intersections
            hits.data_ptr<bool>(),         // [n_rays, n_grids]
            t_sorted.data_ptr<float>(),    // [n_rays, n_grids * 2]
            t_indices.data_ptr<int64_t>(), // [n_rays, n_grids * 2]
            // options
            near_planes.data_ptr<float>(), // [n_rays]
            far_planes.data_ptr<float>(),  // [n_rays]
            step_size,
            cone_angle,
            traverse_steps_limit,
            // outputs
            true,
            device::PackedRaySegmentsSpec(intervals),
            device::PackedRaySegmentsSpec(samples),
            nullptr);  /* terminate_planes */
        
        // second pass to record the segments.
        if (compute_intervals)
            intervals.memalloc_data_from_chunk(true, true);
        if (compute_samples)
            samples.memalloc_data_from_chunk(false, false, true);
        device::traverse_grids_kernel<<<blocks, threads, 0, stream>>>(
            // rays
            n_rays,
            rays_o.data_ptr<float>(),  // [n_rays, 3]
            rays_d.data_ptr<float>(),  // [n_rays, 3]
            nullptr,  /* rays_mask */
            // grids
            n_grids,
            resolution,
            binaries.data_ptr<bool>(), // [n_grids, resx, resy, resz]
            aabbs.data_ptr<float>(),   // [n_grids, 6]
            // sorted intersections
            hits.data_ptr<bool>(),         // [n_rays, n_grids]
            t_sorted.data_ptr<float>(),    // [n_rays, n_grids * 2]
            t_indices.data_ptr<int64_t>(), // [n_rays, n_grids * 2]
            // options
            near_planes.data_ptr<float>(), // [n_rays]
            far_planes.data_ptr<float>(),  // [n_rays]
            step_size,
            cone_angle,
            traverse_steps_limit,
            // outputs
            false,
            device::PackedRaySegmentsSpec(intervals),
            device::PackedRaySegmentsSpec(samples),
            compute_terminate_planes ? terminate_planes.data_ptr<float>() : nullptr);
    }
    
    return {intervals, samples, terminate_planes};
}


std::vector<torch::Tensor> ray_aabb_intersect(
    const torch::Tensor rays_o, // [n_rays, 3]
    const torch::Tensor rays_d, // [n_rays, 3]
    const torch::Tensor aabbs,  // [n_aabbs, 6]
    const float near_plane,
    const float far_plane, 
    const float miss_value)  
{
    DEVICE_GUARD(rays_o);

    int32_t n_rays = rays_o.size(0);
    int32_t n_aabbs = aabbs.size(0);
    int32_t numel = n_rays * n_aabbs;

    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
    int32_t max_threads = 512; 
    int32_t max_blocks = 65535;
    dim3 threads = dim3(min(max_threads, numel));
    dim3 blocks = dim3(min(max_blocks, ceil_div<int32_t>(numel, threads.x)));

    // outputs
    torch::Tensor t_mins = torch::empty({n_rays, n_aabbs}, rays_o.options());
    torch::Tensor t_maxs = torch::empty({n_rays, n_aabbs}, rays_o.options());
    torch::Tensor hits = torch::empty({n_rays, n_aabbs}, rays_d.options().dtype(torch::kBool));

    device::ray_aabb_intersect_kernel<<<blocks, threads, 0, stream>>>(
        // rays
        n_rays,
        rays_o.data_ptr<float>(),  // [n_rays, 3]
        rays_d.data_ptr<float>(),  // [n_rays, 3]
        near_plane,
        far_plane,
        // aabbs
        n_aabbs,
        aabbs.data_ptr<float>(),   // [n_aabbs, 6]
        // outputs
        miss_value,
        t_mins.data_ptr<float>(),   // [n_rays, n_aabbs]
        t_maxs.data_ptr<float>(),   // [n_rays, n_aabbs]
        hits.data_ptr<bool>());     // [n_rays, n_aabbs]

    return {t_mins, t_maxs, hits};
}
